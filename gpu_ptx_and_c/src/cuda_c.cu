#include "hip/hip_runtime.h"
#include "../include/cuda_c.cuh"

#ifdef SHARED

__global__ void matmul_shared(const int* A, const int* B, int* C, const int M, const int N, const int K) {

    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int sy = threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int sx = threadIdx.x;

    extern __shared__ float smem[];
    float *sA = &smem[0];
    float *sB = &smem[blockDim.x*blockDim.x];

    int sum = 0;
    for (int t=0; t<K; t+=blockDim.x) {
        
        if (y<M && sx+t<K) {
            sA[sy*blockDim.x+sx] = A[y*K+(sx+t)];
        } else {
            sA[sy*blockDim.x+sx] = 0;
        }

        if (x<N && sy+t<K) {
            sB[sy*blockDim.x+sx] = B[(sy+t)*N+x];
        } else {
            sB[sy*blockDim.x+sx] = 0;
        }

        __syncthreads();

        for (int k=0; k<blockDim.x; k++) {
            sum += sA[sy*blockDim.x+k]*sB[k*blockDim.x+sx];
        }

        __syncthreads();

    }

    if (y<M && x<N) {
        C[y*N+x] = sum;
    }

}



#else


// Kernel CUDA para multiplicação de matrizes básica
__global__ void matmul_basic(const int* A, const int* B, int* C, const int M, const int N, const int K) {

    // Calcula o índice global da thread na matriz C (coordenada y e x)
    // blockIdx.y e blockIdx.x: índices do bloco atual em ambas as dimensões
    // blockDim.y e blockDim.x: dimensões do bloco (threads por dimensão)
    // threadIdx.y e threadIdx.x: índices da thread dentro do bloco
    int y = blockIdx.y * blockDim.y + threadIdx.y; // Índice da linha na matriz C
    int x = blockIdx.x * blockDim.x + threadIdx.x; // Índice da coluna na matriz C

    // Verifica se a thread está dentro dos limites válidos da matriz C
    if (y < M && x < N) { 
        int sum = 0; // Inicializa o acumulador para calcular C[y][x]

        // Itera sobre os elementos da linha de A e da coluna de B
        for (int k = 0; k < K; k++) {
            // Calcula o produto dos elementos correspondentes de A e B
            // A[y * K + k]: Acessa o elemento da linha y e coluna k em A
            // B[k * N + x]: Acessa o elemento da linha k e coluna x em B
            sum += A[y * K + k] * B[k * N + x];
        }

        // Armazena o resultado na matriz C na posição correspondente
        // C[y * N + x]: Calcula o índice linear da posição (y, x) em C
        C[y * N + x] = sum;
    }
}

#endif